#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * cuNearestNeighbor.cu -- Find the nearest neighbor kernel
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "cuNearestNeighbor2ndPass.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// https://devblogs.nvidia.com/cuda-pro-tip-flush-denormals-confidence/
#define SOFTENING 1e-9f
#define DIM 3

__constant__ int d_tensorVector0[3];
__constant__ int d_tensorVector1[3];
__constant__ int d_tensorVector2[3];

// S has the points that will look for the nearerst neighbor
// P is a gridded representation of the Q points vector
// d3 is the d value cubed. AKA the number of the grids.

__global__
void cuNearestNeighbor2ndPass(float *C, int *S, float *Q, int NQ, int *checkedQInBox, int d, int *neighbor, char *checkOutside) {

  __shared__ float total_nearestDist;
  __shared__ int total_nearestIdx;

  int boxIdToCheck;
  int proccess =  blockIdx.x;
  int stride = gridDim.x;

  int d3 = d*d*d;
  // int d2 = d*d;
  float *q, *c;
  float dx, dy, dz, distSqr, dist, nearestDist;
  int boxId, nearestIdx;


  for(int idx=proccess; idx<NQ; idx+=stride) {
    if(checkOutside[idx] == 1) { 
    // Probably "== 1" is not needed, but since type variable is char, 
    //it was introduced and may be dumped if it operates without it 

      // Could have been 0 as well
      if(threadIdx.x==13) {
        total_nearestDist = 1.000000;
        total_nearestIdx = -1;
      }
      __syncthreads();

      q = Q+(DIM*idx);
      boxId = checkedQInBox[idx];
      nearestDist = 1;        //This is HUGE!

      // Calculate the boxIdToCheck of each thread, depending on its Idx
      int div9 = (int)threadIdx.x/9;
      int mod9 = (int)threadIdx.x%9;
      int div3 = (int)mod9/3;
      int mod3 = (int)mod9%3;
      boxIdToCheck = boxId + d_tensorVector0[mod3] + d_tensorVector1[div3] + d_tensorVector2[div9]; 

      if(boxIdToCheck < d3 && boxIdToCheck >=0) {
        for(int S_num=S[boxIdToCheck]; S_num<S[boxIdToCheck+1]; S_num+=3){
          c = C+(S_num);
          dx = q[0] - c[0];
          dy = q[1] - c[1];
          dz = q[2] - c[2];
          distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
          dist = sqrtf(distSqr);
          if(dist<nearestDist){
          	nearestDist = dist;
          	nearestIdx = S_num;
          } 
        } // end of for(int S_num=0; S_num<SDim[boxIdToCheck]; S_num++)
        
        if(nearestDist<total_nearestDist) {
          atomicExch(&total_nearestDist, nearestDist);
          atomicExch(&total_nearestIdx, nearestIdx);
        }
      }

      if(threadIdx.x==13)
        neighbor[idx] = total_nearestIdx;  
    }        
  } // end of  for(int P_num=0; P_num<P_size[i]; P_num++)

}