#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * hashing3D.cu -- Hashing the 3D space into d*d*d boxes
 * 
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "hashing3D.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DIM 3

void cumsum(int *array, int index) {
    if(index <= 0) return;
    cumsum(array, index -1);
    array[index] += array[index - 1];
}

__global__
void cuInitBoxCount(int *d_boxCount, int n) {

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i=process; i<n; i+=stride){
    d_boxCount[i]=0;
  }
}

__global__
void cuFindBelongsToBox (float *v, int N, int d, int *belongsToBox, int *positionWithinBox, int *boxSize){

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int d2 = d * d;
  int d3 = d * d * d;
  float x, y, z;

  for(int n=process; n<N; n+=stride){
      x = v[n*DIM];
      y = v[n*DIM+1];
      z = v[n*DIM+2];
      belongsToBox[n] = (int)(x*d) +(int)(y*d)*d +(int)(z*d)*d2; //= boxId;
      positionWithinBox[n] = atomicAdd(&boxSize[belongsToBox[n]], 1);
      // printf("> v[%d]: belongsToBox %d, positionWithinBox=%d, boxSize=%d\n",n,belongsToBox[n],positionWithinBox[n],boxSize[belongsToBox[n]]);
    }
}


__global__
void cuPrefixSum (int *array, int size){
// CUDA implementation of the Exclusive Prefix Sum Algorithm for array boxStart
// Reference: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf

  extern __shared__ int temp[];

  int process = threadIdx.x;
  int offset = 1;

  // AS IS, IT WON'T WORK FOR d > 3
  if(size>1024) printf("size>1024: PROBLEM\n");

	if (process<size/2) {
	  temp[2*process]   = array[2*process];
	  temp[2*process+1] = array[2*process+1];

	  for(int n = size>>1; n > 0; n >>= 1)
	  {
	  	__syncthreads();

	  	if(process < n)
	  	{
	  		int ai = offset*(2*process+1)-1;
	  		int bi = offset*(2*process+2)-1;

	  		temp[bi] += temp[ai];
	  	}

	  	offset *= 2;
	  }

	  if(process == 0) {
	  	array[size] = temp[size-1];
	    temp[size-1] = 0;
	  }

	  for(int n = 1; n < size; n *= 2)
	  {
	  	offset >>= 1;
	  	__syncthreads();

	  	if(process < n)
	  	{
	  		int ai = offset*(2*process+1)-1;
	  		int bi = offset*(2*process+2)-1;

	  		float t   = temp[ai];
	  		temp[ai]  = temp[bi];
	  		temp[bi] += t;
	  	}
	  }

	  __syncthreads();

	  array[2*process] = temp[2*process];
	  array[2*process+1] = temp[2*process+1];

	  //printf("boxStart[%d]=%d\n",2*process,array[2*process]);
	  //printf("boxStart[%d]=%d\n",2*process+1,array[2*process+1]);
	}
}

__global__
void cuRearrangeV (float *v, float *newV, int N, int d, int *belongsToBox, int *newBelongsToBox, int *positionWithinBox, int *boxStart){

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int position;

  for(int n=process; n<N; n+=stride){
    position   = boxStart[belongsToBox[n]] + positionWithinBox[n];
    newV[DIM*position]   = v[DIM*n];
    newV[DIM*position+1] = v[DIM*n+1];
    newV[DIM*position+2] = v[DIM*n+2];
    newBelongsToBox[position] = belongsToBox[n];
    }
}


int hashing3D(float *v, float **d_v, size_t vSize, int N, int d, int **vPartsStart, int **d_vPartsStart,
              int **vBelongsToBox, int **d_vBelongsToBox, size_t numberOfBlocks, size_t threadsPerBlock)
{

  int d3 = d*d*d;
  int *belongsToBox, *d_newBelongsToBox, *d_belongsToBox, *d_positionWithinBox, *boxStart, *d_boxStart;
  hipError_t err;
  float *d_newV;

  size_t belongsToBoxSize = N*sizeof(int);
  size_t boxStartSize     = (d3+1)*sizeof(int);

  CUDA_CALL(hipMalloc(&d_belongsToBox, belongsToBoxSize));
  CUDA_CALL(hipMalloc(&d_newBelongsToBox, belongsToBoxSize));
  CUDA_CALL(hipMalloc(&d_positionWithinBox, belongsToBoxSize));
  CUDA_CALL(hipMalloc(&d_boxStart, boxStartSize));
  CUDA_CALL(hipMalloc(&d_newV, vSize));

  belongsToBox = (int *)malloc(belongsToBoxSize);
  if(belongsToBox == NULL) {
    printf("Error allocating belongsToBox\n");
    exit(1);
  }
  boxStart = (int *)malloc(boxStartSize);
  if(boxStart == NULL) {
    printf("Error allocating boxStart\n");
    exit(1);
  }  
  // printf("tr:%zu, bl:%zu\n",threadsPerBlock, numberOfBlocks);
  cuInitBoxCount<<<threadsPerBlock, numberOfBlocks>>>(d_boxStart, d3+1);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  CUDA_CALL(hipDeviceSynchronize());

  cuFindBelongsToBox<<<threadsPerBlock, numberOfBlocks>>>
    (*d_v, N, d, d_belongsToBox, d_positionWithinBox, d_boxStart+1);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  CUDA_CALL(hipDeviceSynchronize());

  size_t maxNumOfThreads = 1024;
  
  // printf(" ==== Prefix Sum ==== \n");

  // As is, IT WON'T WORK FOR d > 3
  // We will have to try an implementation with more than 1 blocks, 
  // in order to yeild maximum performonce 
/*  cuPrefixSum<<<1, maxNumOfThreads, boxStartSize>>>
    (d_boxStart, d3);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  CUDA_CALL(hipDeviceSynchronize());
*/

  CUDA_CALL(hipMemcpy(boxStart, d_boxStart, boxStartSize, hipMemcpyDeviceToHost));

  cumsum(boxStart,d3);
  
/*  for(int i=0;i<d3;i++)
    printf("%d: %d, ",i, boxStart[i]);
  printf("\n");
*/
  CUDA_CALL(hipMemcpy(d_boxStart, boxStart, boxStartSize, hipMemcpyHostToDevice));

  cuRearrangeV<<<threadsPerBlock, numberOfBlocks>>>
    (*d_v, d_newV, N, d, d_belongsToBox, d_newBelongsToBox, d_positionWithinBox, d_boxStart);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(v, d_newV, vSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(*d_v));
  CUDA_CALL(hipMemcpy(boxStart, d_boxStart, boxStartSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(belongsToBox, d_newBelongsToBox, belongsToBoxSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(d_belongsToBox));
  *d_v = d_newV;

  *vPartsStart = boxStart;
  *d_vPartsStart = d_boxStart;
  *vBelongsToBox = belongsToBox;
  *d_vBelongsToBox = d_newBelongsToBox;

  CUDA_CALL(hipFree(d_positionWithinBox));
    
  return 0;
}

int hashing3D(float *v, float **d_v, size_t vSize, int N, int d, int **vPartsStart, int **d_vPartsStart,
              size_t numberOfBlocks, size_t threadsPerBlock) {

  int *belongsToBox, *d_belongsToBox;
  int ret = hashing3D(v,d_v,vSize,N,d,vPartsStart,d_vPartsStart,&belongsToBox,
		      &d_belongsToBox,numberOfBlocks,threadsPerBlock);
  CUDA_CALL(hipFree(d_belongsToBox));
  free(belongsToBox);
  return ret;
}