#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * hashing3D.cu -- Hashing the 3D space into d*d*d boxes
 * 
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "hashing3D.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DIM 3

__global__
void hashingKernel(float *v, int N, int d, int boxIdx, float **pointersToBoxPoints, int *size_boxPoints)
{
	int proccess = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	int d2 = d * d;
	float d_inv = (float)1/d;

	float x, y, z;

	for (int i = proccess; i < N; i += stride)
	{
		x = v[i*DIM];
		y = v[i*DIM+1];
		z = v[i*DIM+2];

		if(boxIdx == ((int)(x/d_inv) * 1) + ((int)(y/d_inv) * d) + ((int)(z/d_inv) * d2)) {
			int pos = atomicAdd(size_boxPoints, 1);
			pointersToBoxPoints[pos] = &v[i*DIM];
		}
	}
}


int hashing3D(float *v, int N, int d, float ****boxes, int **boxesSizes,
              size_t numberOfBlocks, size_t threadsPerBlock)
{
	int d3 = d * d * d;

	float ***boxes_temp, **ptr, **pointersToBoxPoints;
	int *boxesSizes_temp, *size_boxPoints;
	size_t size, count, max_size_boxPoints;

	CUDA_CALL(hipMallocManaged(&boxesSizes_temp, d3 * sizeof(int)) );

	size = d3 * sizeof(float **) + N * sizeof(float*);
	// https://www.geeksforgeeks.org/dynamically-allocate-2d-array-c/

	CUDA_CALL(hipMallocManaged(&boxes_temp, size));

	CUDA_CALL(hipMallocManaged(&size_boxPoints, 1*sizeof(int))); 

    // Alocating memory for float** to hold the pointers pointing the box points of boxIdx, as found by the hashingKernel
	max_size_boxPoints = N * sizeof(float**);

	CUDA_CALL(hipMallocManaged(&pointersToBoxPoints, max_size_boxPoints)); 
    
	// counter to hold the sum of points mapped to all boxIdxs < curent i
	count = 0;

	// Reserving the first d3 positions of boxes_temp for the boxes_temp[i] double pointers
	ptr =(float **)boxes_temp + d3;

    for(int i=0;i<d3;i++) {

		*size_boxPoints = 0;
		
		hashingKernel<<<numberOfBlocks, threadsPerBlock>>>
            (v, N, d, i, pointersToBoxPoints, size_boxPoints);
        
		hipDeviceSynchronize();
		
		// size_boxPoints is the number of points that are mapped to boxIdx = i, so
		boxesSizes_temp[i] = (*size_boxPoints);
		
		// boxes_temp[i] = boxes_temp + sum of points mapped to all boxIdxs < i
		boxes_temp[i] = ptr + count;
		
		// Post incrementing the counter count, so that it gives the correct offset for the next iteration
		count += (*size_boxPoints);
		
		for(int boxPoints = 0; boxPoints < (*size_boxPoints); boxPoints++)
			// Placing each pointer of each box's point into the appropriate positions of boxes_temp
			boxes_temp[i][boxPoints] = pointersToBoxPoints[boxPoints];
	}

	// Access of boxes array: boxes[d3][N][0,1,2]; actually it's boxes[d3][boxesSizes[d3]][0,1,2] with boxesSizes[d3] having a sum of N points
	*boxes = boxes_temp;
	*boxesSizes = boxesSizes_temp;
    
    return 0;
}
