#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * hashing3D.cu -- Hashing the 3D space into d*d*d boxes
 * 
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "hashing3D.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DIM 3

__global__
void cuFindBelongsToBox (float *v, int N, int d, int *belongsToBox, int *positionWithinBox, int *boxSize){

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int d2 = d * d;
  int d3 = d * d * d;
  float x, y, z;

  for(int n=process; n<d3; n+=stride)
  	  boxSize[n]=0;

  __syncthreads();

  for(int n=process; n<N; n+=stride){
      x = v[n*DIM];
      y = v[n*DIM+1];
      z = v[n*DIM+2];
      belongsToBox[n] = (int)(x*d) +(int)(y*d)*d +(int)(z*d)*d2; //= boxId;
      positionWithinBox[n] = atomicAdd(&boxSize[belongsToBox[n]], 1);
      // printf("> v[%d]: belongsToBox %d, positionWithinBox=%d, boxSize=%d\n",n,belongsToBox[n],positionWithinBox[n],boxSize[belongsToBox[n]]);
    }
}


__global__
void cuPrefixSum (int *array, int size){
// CUDA implementation of the Exclusive Prefix Sum Algorithm for array boxStart
// Reference: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf

  extern __shared__ int temp[];

  int process = threadIdx.x;
  int offset = 1;

  // AS IS, IT WON'T WORK FOR d > 3
  if(size>1024) printf("size>1024: PROBLEM\n");

if (process<size/2) {
	  temp[2*process]   = array[2*process];
	  temp[2*process+1] = array[2*process+1];

	  for(int n = size>>1; n > 0; n >>= 1)
	  {
	  	__syncthreads();

	  	if(process < n)
	  	{
	  		int ai = offset*(2*process+1)-1;
	  		int bi = offset*(2*process+2)-1;

	  		temp[bi] += temp[ai];
	  	}

	  	offset *= 2;
	  }

	  if(process == 0) {
	  	array[size] = temp[size-1];
	    temp[size-1] = 0;
	  }

	  for(int n = 1; n < size; n *= 2)
	  {
	  	offset >>= 1;
	  	__syncthreads();

	  	if(process < n)
	  	{
	  		int ai = offset*(2*process+1)-1;
	  		int bi = offset*(2*process+2)-1;

	  		float t   = temp[ai];
	  		temp[ai]  = temp[bi];
	  		temp[bi] += t;
	  	}
	  }

	  __syncthreads();

	  array[2*process] = temp[2*process];
	  array[2*process+1] = temp[2*process+1];

	  //printf("boxStart[%d]=%d\n",2*process,array[2*process]);
	  //printf("boxStart[%d]=%d\n",2*process+1,array[2*process+1]);
	}
}

__global__
void cuRearrangeV (float *v, int N, int d, int *belongsToBox, int *positionWithinBox, int *boxStart){

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int position;

  for(int n=process; n<N; n+=stride){
    position   = boxStart[belongsToBox[n]] + positionWithinBox[n];
    v[DIM*n]   = atomicExch( &v[DIM*position], v[DIM*n]);
    v[DIM*n+1] = atomicExch( &v[DIM*position+1], v[DIM*n+1]);
    v[DIM*n+2] = atomicExch( &v[DIM*position+2], v[DIM*n+2]);
    belongsToBox[n] = atomicExch( &belongsToBox[position], belongsToBox[n]);
    }
}


int hashing3D(float *v, float *d_v, size_t vSize, int N, int d, int **vPartsStart, int **d_vPartsStart,
              int **vBelongsToBox, int **d_vBelongsToBox, size_t numberOfBlocks, size_t threadsPerBlock)
{

  int d3 = d*d*d;
  int *belongsToBox, *d_belongsToBox, *d_positionWithinBox, *boxStart, *d_boxStart;
  hipError_t err;
  
  size_t belongsToBoxSize = N*sizeof(int);
  size_t boxStartSize     = (d3+1)*sizeof(int);

  CUDA_CALL(hipMalloc(&d_belongsToBox, belongsToBoxSize));
  CUDA_CALL(hipMalloc(&d_positionWithinBox, belongsToBoxSize));
  CUDA_CALL(hipMalloc(&d_boxStart, boxStartSize));

  belongsToBox = (int *)malloc(belongsToBoxSize);
  if(belongsToBox == NULL) {
    printf("Error allocating belongsToBox\n");
    exit(1);
  }
  boxStart = (int *)malloc(boxStartSize);
  if(boxStart == NULL) {
    printf("Error allocating boxStart\n");
    exit(1);
  }  
  // printf("tr:%zu, bl:%zu\n",threadsPerBlock, numberOfBlocks);

  cuFindBelongsToBox<<<threadsPerBlock, numberOfBlocks>>>
    (d_v, N, d, d_belongsToBox, d_positionWithinBox, d_boxStart);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  CUDA_CALL(hipDeviceSynchronize());

  size_t maxNumOfThreads = 1024;
  
  // printf(" ==== Prefix Sum ==== \n");

  // As is, IT WON'T WORK FOR d > 3
  // We will have to try an implementation with more than 1 blocks, 
  // in order to yeild maximum performonce 
  cuPrefixSum<<<1, maxNumOfThreads, boxStartSize>>>
    (d_boxStart, d3);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  CUDA_CALL(hipDeviceSynchronize());

  cuRearrangeV<<<threadsPerBlock, numberOfBlocks>>>
    (d_v, N, d, d_belongsToBox, d_positionWithinBox, d_boxStart);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(v, d_v, vSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(boxStart, d_boxStart, boxStartSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(belongsToBox, d_belongsToBox, belongsToBoxSize, hipMemcpyDeviceToHost));

  *vPartsStart = boxStart;
  *d_vPartsStart = d_boxStart;
  *vBelongsToBox = belongsToBox;
  *d_vBelongsToBox = d_belongsToBox;

  CUDA_CALL(hipFree(d_positionWithinBox));
    
  return 0;
}

int hashing3D(float *v, float *d_v, size_t vSize, int N, int d, int **vPartsStart, int **d_vPartsStart,
              size_t numberOfBlocks, size_t threadsPerBlock) {

  int *belongsToBox, *d_belongsToBox;
  int ret = hashing3D(v,d_v,vSize,N,d,vPartsStart,d_vPartsStart,&belongsToBox,
		      &d_belongsToBox,numberOfBlocks,threadsPerBlock);
  CUDA_CALL(hipFree(d_belongsToBox));
  free(belongsToBox);
  return ret;
}