#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * cpuValidation.c -- cpuValidation function for the NNs of Q as 
 * 					  calculated via CUDA
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/
#include "cpuValidation.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define DIM 3

int cpuValidation(float *Q, int NQ, float *C, int NC, int *results, char verboseFlag)
{
  float NNdist, dist;
  int NNidx;
  int flag = 0;

  float xQ, yQ, zQ;
  float xC, yC, zC;

  if(verboseFlag==1)
    printf("\n\n ====== Begining validation of results ======\n\n");

  for(int i = 0; i < NQ; i++) {
    NNdist=1.000000;
    xQ = Q[i * DIM + 0];
    yQ = Q[i * DIM + 1];
    zQ = Q[i * DIM + 2];

    for(int j = 0; j < NC; j++) {
      dist = 0.000000;
      xC = C[j * DIM + 0];
      yC = C[j * DIM + 1];
      zC = C[j * DIM + 2];

      dist = (xQ-xC)*(xQ-xC) + (yQ-yC)*(yQ-yC) + (zQ-zC)*(zQ-zC);
      dist = sqrtf(dist);

      if(dist<NNdist) {
	NNdist = dist;
	NNidx = j;
      }			
    } // End of going through all C for the one q

    if(results[i] != NNidx) {
      printf("     ! ! ! VALIDATION FAILED ! ! !\n");
      printf("-> On Q[%d]: (%1.4f, %1.4f, %1.4f)\n",i, xQ, yQ, zQ);
      if(results[i] == -1)
        printf("\nAlgorithm did not manage to locate a neighbor within the Primary nor the Secondary Candidates\n\n\n");
      else {
        printf("Algorithm found C[%d] as the NN, while in fact it was C[%d].\n", results[i], NNidx);
        printf("      (%1.4f, %1.4f, %1.4f)                 (%1.4f, %1.4f, %1.4f)\n\n",
  	     C[results[i]*DIM],C[results[i]*DIM+1],C[results[i]*DIM+2],
  	     C[NNidx*DIM],C[NNidx*DIM+1],C[NNidx*DIM+2]);
      }
      flag = 1;
      if(verboseFlag==0) // If verboseFlag is enabled quit as soon as you find the first miscalculated NN
      	return flag;
    }

  } // End of going through all Q;

  if(flag==0) // If it reached here with flag == 0 then it has found no error
    printf("     ! ! ! VALIDATION SUCCEEDED ! ! !\n\n");
      
  return flag;
}