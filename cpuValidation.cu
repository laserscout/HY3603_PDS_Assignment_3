#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * cpuValidation.c -- cpuValidation function for the NNs of Q as 
 * 					  calculated via CUDA
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/
#include "cpuValidation.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define DIM 3

void cpuValidation(float *Q, int NQ, float *C, int NC, int *results)
{
	float NNdist, dist;
	int NNidx;

	float xQ, yQ, zQ;
	float xC, yC, zC;
    
    printf("\n\n ====== Begining validation of results ======\n\n");

	for(int i = 0; i < NQ; i++) {
		NNdist=1.000000;
		xQ = Q[i * DIM + 0];
		yQ = Q[i * DIM + 1];
		zQ = Q[i * DIM + 2];

		for(int j = 0; j < NC; j++) {
			dist = 0.000000;
			xC = C[j * DIM + 0];
			yC = C[j * DIM + 1];
			zC = C[j * DIM + 2];

			dist = (xQ-xC)*(xQ-xC) + (yQ-yC)*(yQ-yC) + (zQ-zC)*(zQ-zC);
			dist = sqrtf(dist);

			if(dist<NNdist) {
			  NNdist = dist;
			  NNidx = j;
			}			
		}

		if(results[i]/3 != NNidx) {

			printf("     ! ! ! VALIDATION FAILED ! ! !\n");
            printf("-> On Q[%d]: (%1.4f, %1.4f, %1.4f)\n",i, xQ, yQ, zQ);
			printf("Algorithm found C[%d] as the NN, while in fact it was C[%d].\n\n", results[i]/3, NNidx);
		}

	}
    printf("\n");
}