#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * cuNearestNeighbor.cu -- Find the nearest neighbor kernel
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "cuNearestNeighbor.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// https://devblogs.nvidia.com/cuda-pro-tip-flush-denormals-confidence/
#define SOFTENING 1e-9f
#define DIM 3


// S has the points that will look for the nearerst neighbor
// P is a gridded representation of the Q points vector
// d3 is the d value cubed. AKA the number of the grids.

__global__
void cuNearestNeighbor(float **S, int *SDim, float *Q, int *checkQInBox, int d, float **neighbor) {

  int proccess = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int d3 = d*d*d;
  int d2 = d*d;
  float *q, *c, *nearestPoint;
  float dx, dy, dz, distSqr, dist, nearestDist;
  float sqrDx[3], sqrDy[3], sqrDz[3];
  sqrDx[0]=0; sqrDy[0]=0; sqrDz[0]=0;
  int boxId, gridX, gridY, gridZ, temp;


  for(int idx=proccess; idx<d3; idx+=stride) {
    q = Q+(DIM*idx);
    boxId = checkQInBox[idx];
    nearestDist = 1;        //This is HUGE!
    for(int S_num=0; S_num<SDim[boxId]; S_num++){
      c  = S[boxId] + (DIM * S_num);  //check distance of c anf q;
      dx = q[0] - c[0];
      dy = q[1] - c[1];
      dz = q[2] - c[2];
      distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      dist = sqrtf(distSqr);
      if(dist<nearestDist){
	nearestDist = dist;
	nearestPoint = c;
      } // !!Try two nops here as an else???
    } // end of for(int S_num=0; S_num<SDim[boxId]; S_num++)
    neighbor[idx]=nearestPoint;

    // These are the XYZ coordinates of the grid
    gridZ = boxId / d2;
    temp  = boxId % d2;
    gridY = temp / d;
    gridX = temp % d;

    // Now calculate the distance of the point from:
    // the 8 verteces of the grid cube
    // the 12 edges
    // and the 6 faces

    dx       = q[0] - gridX;
    sqrDx[1] = dx*dx;
    sqrDx[2] = (d-dx)*(d-dx);
    dy       = q[1] - gridY;
    sqrDy[1] = dy*dy;
    sqrDy[2] = (d-dy)*(d-dy);
    dz       = q[2] - gridZ;
    sqrDz[1] = dz*dz;
    sqrDz[2] = (d-dz)*(d-dz);
    // Reminder that sqrD(xyz)[0] = 0
      
    for(int zi=0; zi<3; zi++){
      for(int yi=0; yi<3; yi++){
	for(int xi=0; xi<3; xi++){
	  distSqr = sqrDx[xi] + sqrDy[yi] + sqrDz[zi];
	  //cbrtf(distSqr);
	}
      }
    }
      
  } // end of  for(int P_num=0; P_num<P_size[i]; P_num++)
}

// if(dx<nearestDist) { // It's near the left side x face
//   //code
//   if(cbrtf(dx*dx+dy*dy)<nearestDist) { // left x, bottom y edge
//     //code
//   }
//   else if(cbrtf(dx*dx+
// 		}
// 	  else if(d-dx<nearestDist) { // It's near the right side x face
// 	  }
      
// 	  if(dy<nearestDist) { // Left y face
// 	  }
// 	  else if(d-dy<nearestDist) { // Right y face
// 	  }
      
// 	  if(dz<nearestDist) { // Left z face
// 	  }
// 	  else if(d-dz<nearestDist) { // Right z face
// 	  }

// S[boxid][i][0,1,2]

// // secont way

// S_acutal[boxid][3*i+"0, 1, 2"]

// S[ 3*( boxid*S_size[boxid] + i ) + "0,1,2"]