#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * cuNearestNeighbor.cu -- Find the nearest neighbor kernel
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include "cuNearestNeighbor.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// https://devblogs.nvidia.com/cuda-pro-tip-flush-denormals-confidence/
#define SOFTENING 1e-9f
#define DIM 3


// S has the points that will look for the nearerst neighbor
// P is a gridded representation of the Q points vector
// d3 is the d value cubed. AKA the number of the grid boxes.

__global__
void cuNearestNeighbor(float *C, int *S, float *Q, int NQ, int *checkQInBox, int d, int *neighbor, char *checkOutside) {

  int process = threadIdx.x + blockIdx.x * blockDim.x;
  // int stride = blockDim.x * gridDim.x;

  // int d3 = d*d*d;
  int d2 = d*d;
  float invd = 1/(float)d;
  float *q, *c;
  float q_x, q_y, q_z;
  float dx, dy, dz, distSqr, dist, nearestDist, gridX, gridY, gridZ;
  int boxId, temp, nearestIdx;

  // for(int idx=process; idx<NQ; idx+=stride) {
    q = Q+(DIM*process);
    q_x = q[0];
    q_y = q[1];
    q_z = q[2];

    boxId = checkQInBox[process];
    nearestDist = 1;        // This is HUGE!
    nearestIdx = -1;        // Error checking value
    // printf("q[%d]:%1.4f, %1.4f, %1.4f | Belongs to %d\n",process,q[0],q[1],q[2],boxId);
    for(int S_num=S[boxId]; S_num<S[boxId+1]; S_num++){
      c = C+(S_num*DIM);
      dx = q_x - c[0];
      dy = q_y - c[1];
      dz = q_z - c[2];
      distSqr = dx*dx + dy*dy + dz*dz;
      dist = sqrtf(distSqr);
      if(dist<nearestDist){
      	nearestDist = dist;
      	nearestIdx = S_num;
      }
    } // end of for(int S_num=0; S_num<SDim[boxId]; S_num++)
    neighbor[process]=nearestIdx;
   
    // These are the XYZ coordinates of the grid box
    gridZ = (boxId / d2) * invd;
    temp  = boxId % d2;
    gridY = (temp / d) * invd;
    gridX = (temp % d) * invd;

    // Now calculate the distances of the point from the 6 faces
    dx = q[0] - gridX;
    dy = q[1] - gridY;
    dz = q[2] - gridZ;
    
    /*
    Here remove checkOutside and perform the 26 checks as in older version 
    (nearestKernel it was I think) then hold the 26 results in a local array 
    and merge this function with 2nd Pass and this way we are good to go 
    with a single pass ! ! ! And reduced checks ! ! !
    */
    if( (dx)<nearestDist || (invd-dx)<nearestDist ||
      	(dy)<nearestDist || (invd-dy)<nearestDist ||
      	(dz)<nearestDist || (invd-dz)<nearestDist  )
      checkOutside[process]=1;
    else
      checkOutside[process]=0;      
      
  // } // end of  for(int P_num=0; P_num<P_size[i]; P_num++)
}
