/**********************************************************************
 *
 * main.cu -- main function for the NN in CUDA
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "cuRandFloat.h"
#include "hashing3D.h"
#include "cpuValidation.h"
#include "cuNearestNeighbor.h"
#include "cuNearestNeighbor2ndPass.h"

#define DIM 3

int main (int argc, char *argv[]) {

  float *Q, *C, *d_Q, *d_C;
  size_t QSize, CSize;
  int *S, *d_S, *P, *d_P;
  int NC, NQ, d;
  int SDim;
  hipError_t err;
  char verboseFlag = 0;
  char noValidationFlag = 0;

  if (argc < 4) {
    printf("Usage: %s [flags] arg1 arg2 arg3\n  where NC=2^arg1, NQ=2^arg2 and d=2^arg3\n",
	   argv[0]);
    exit(1);
  }

  for(int i=1; i<argc; i++) {
    if (strcmp(argv[i], "-v") == 0)
      {                 
        verboseFlag = 1; // use only with small NC NQ and d
      }
    if (strcmp(argv[i], "--novalidation") == 0)
      {                 
        noValidationFlag = 1; // Do not run the slow validation in the end
      }
    if (strncmp(argv[i], "-", 1) != 0) {
      NC = 1<<atoi(argv[i]);
      NQ = 1<<atoi(argv[i+1]);
      d  = 1<<atoi(argv[i+2]);
      break;
    }
      
  }
  
  size_t threadsPerBlock, warp;
  size_t numberOfBlocks, multiP;

  int deviceId;
  hipDeviceProp_t props;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);

  warp = props.warpSize;
  multiP = props.multiProcessorCount;
  threadsPerBlock = 8*warp;
  numberOfBlocks  = 5*multiP;

  randFloat(&Q, &d_Q, NQ);
  QSize = DIM * NQ * sizeof(float);
  randFloat(&C, &d_C, NC);
  CSize = DIM * NC * sizeof(float);
  CUDA_CALL(hipDeviceSynchronize());

  if(verboseFlag == 1) {
    /* Show result */
    printf(" ======Q vector====== \n");
    for(int i = 0; i < NQ ; i++){
      for (int d=0; d<DIM; d++)
	printf("%1.4f ", Q[i*DIM+d]);
      printf("\n");
    }
    printf(" ======C vector====== \n");
    for(int i = 0; i < NC ; i++){
      for (int d=0; d<DIM; d++)
	printf("%1.4f ", C[i*DIM+d]);
      printf("\n");
    }
  }

  hipEvent_t startOfHashing, startOfFirstRun, startOfSecondRun, stop;
  hipEventCreate(&startOfHashing);
  hipEventCreate(&startOfFirstRun);
  hipEventCreate(&startOfSecondRun);
  hipEventCreate(&stop);

  hipEventRecord(startOfHashing);

  // Hashing C into d*d*d boxes
  hashing3D(C, d_C, CSize, NC, d, &S, &d_S, 
  		numberOfBlocks, threadsPerBlock);

  int *QBoxIdToCheck, *d_QBoxIdToCheck;
  hashing3D(Q, d_Q, QSize, NQ, d, &P, &d_P, &QBoxIdToCheck, &d_QBoxIdToCheck,
	    numberOfBlocks, threadsPerBlock);

  if(verboseFlag == 1){
    /* Show result */
    printf("\nd=%d\n\n",d);
    printf(" ====new Q vector==== \n");
    for(int i = 0; i < NQ ; i++){
      for (int d=0; d<DIM; d++)
	printf("%1.4f ", Q[i*DIM+d]);
      printf("| Belongs to box:%d\n",QBoxIdToCheck[i]);
    }
    printf(" ======S vector====== \n");
    for(int boxid=0;boxid<d*d*d;boxid++){
      SDim = S[boxid+1]-S[boxid];
      printf("Box%d size=%d\n", boxid, SDim);
      for(int i = S[boxid] ; i < S[boxid+1] ; i++){
        for (int d=0; d<DIM; d++)
          printf("%1.4f ", C[ i*DIM +d ]);
        printf("\n");
      }
    }
  }

  hipEventRecord(startOfFirstRun);

  int *neighbor, *d_neighbor;
  char *d_checkOutside;
  size_t neighborSize = NQ * sizeof(int);
  size_t checkOutsideSize = NQ * sizeof(char);
  
  CUDA_CALL(hipMalloc(&d_neighbor,neighborSize));
  neighbor = (int *)malloc(neighborSize);
  if(neighbor == NULL) {
    printf("Error allocating neighbor");
    exit(1);
  }

  CUDA_CALL(hipMalloc(&d_checkOutside,checkOutsideSize));
  
  hipEventRecord(startOfSecondRun);

  cuNearestNeighbor<<<numberOfBlocks, threadsPerBlock>>>
    (d_C,d_S,d_Q,NQ,d_QBoxIdToCheck,d,d_neighbor,d_checkOutside);

  hipEventRecord(stop);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }
  
  CUDA_CALL(hipDeviceSynchronize());
    
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, startOfHashing, startOfFirstRun);
  printf("Duration of Q and C hashing: %1.6fms\n",milliseconds);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, startOfFirstRun, startOfSecondRun);
  printf("Duration of the first run of the kernel: %1.6fms\n",milliseconds);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, startOfSecondRun, stop);
  printf("Duration of the second run of the kernel: %1.6fms\n",milliseconds);


  CUDA_CALL(hipMemcpy(neighbor, d_neighbor, neighborSize, hipMemcpyDeviceToHost));
  
  if(verboseFlag == 1) {
    printf(" ==== Neighbors! ==== \n");
    for(int i = 0; i < NQ ; i++)
    	printf("> Q[%d] -> C[%d]\n",i,neighbor[i]);
  }

  cuNearestNeighbor2ndPass<<<numberOfBlocks*10, 27>>>
    (d_C,d_S,d_Q,NQ,d_QBoxIdToCheck,d,d_neighbor,d_checkOutside);

  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }

  
  if(verboseFlag == 1) {
    printf(" ==== Neighbors! ==== \n");
    for(int i = 0; i < NQ ; i++)
    	printf(">> Q[%d] -> C[%d]\n",i,neighbor[i]);
  }

  if(noValidationFlag==0) {
    CUDA_CALL(hipMemcpy(neighbor, d_neighbor, neighborSize, hipMemcpyDeviceToHost));
    /* Validating the NN results */
    cpuValidation(Q, NQ, C, NC, neighbor, verboseFlag);
  }
  
  /* Cleanup */
  CUDA_CALL(hipFree(d_Q));
  CUDA_CALL(hipFree(d_C));
  CUDA_CALL(hipFree(d_S));
  CUDA_CALL(hipFree(d_QBoxIdToCheck));
  free(Q);
  free(C);
  free(S);
  free(QBoxIdToCheck);
  
  return 0;
}