/**********************************************************************
 *
 * main.cu -- main function for the NN in CUDA
 *
 * Frank Blanning <frankgou@auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include "cuNearestNeighborHelper.h"
#include "cuRandFloat.h"
#include "hashing3D.h"
#include "cuNearestNeighbor.h"

#define DIM 3

int main (int argc, char *argv[]) {

  float *Q, *C, *d_Q, *d_C;
  size_t QSize, CSize;
  float **S, **d_S, **P, **d_P;
  int NC, NQ, d;
  int *SDim, *d_SDim, *PDim, *d_PDim;
  hipError_t err;

  if (argc != 4) {
    printf("Usage: %s arg1 arg2 arg3\n  where NC=2^arg1, NQ=2^arg2 and d=2^arg3\n",
	   argv[0]);
    exit(1);
  }

  NC = 1<<atoi(argv[1]);
  NQ = 1<<atoi(argv[2]);
  d  = 1<<atoi(argv[3]);
  
  size_t threadsPerBlock, warp;
  size_t numberOfBlocks, multiP;

  int deviceId;
  hipDeviceProp_t props;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);

  warp = props.warpSize;
  multiP = props.multiProcessorCount;
  threadsPerBlock = 8*warp;
  numberOfBlocks  = 5*multiP;

  randFloat(&Q, &d_Q, NQ);
  randFloat(&C, &d_C, NC);
  hipDeviceSynchronize();

  QSize = DIM * NQ * sizeof(float);
  if((Q = (float *)malloc(QSize))==NULL) {
    printf("Malloc error Q\n");
    exit(1);
  }
  CSize = DIM * NQ * sizeof(float);
  if((C = (float *)malloc(CSize))==NULL) {
    printf("Malloc error C\n");
    exit(1);
  }

  CUDA_CALL(hipMemcpy(Q, d_Q, QSize, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(C, d_C, CSize, hipMemcpyDeviceToHost));

  /* Show result */
  printf(" ======Q vector====== \n");
  for(int i = 0; i < NQ ; i++){
    for (int d=0; d<DIM; d++)
      printf("%1.4f ", Q[i*DIM+d]);
    printf("\n");
  }
  printf(" ======C vector====== \n");
  for(int i = 0; i < NC ; i++){
    for (int d=0; d<DIM; d++)
      printf("%1.4f ", C[i*DIM+d]);
    printf("\n");
  }

  // Hashing C into d*d*d boxes
  hashing3D(C, d_C, CSize, NC, d, &S, &d_S, &SDim, &d_SDim, numberOfBlocks, threadsPerBlock);

  int *QBoxIdToCheck, *d_QBoxIdToCheck;
  hashing3D(Q, d_Q, QSize, NQ, d, &P, &d_P, &PDim, &d_PDim, &QBoxIdToCheck, &d_QBoxIdToCheck,
	    numberOfBlocks, threadsPerBlock);

  /* Show result */
  printf("\nd=%d\n\n",d);
  printf(" ======S vector====== \n");
  for(int boxid=0;boxid<d*d*d;boxid++){
    printf("Box%d size=%d\n", boxid, SDim[boxid]);
      for(int i = 0; i < SDim[boxid] ; i++){
        for (int d=0; d<DIM; d++)
          printf("%1.4f ", S[boxid][i*DIM +d]);
        printf("\n");
      }
  }

  float **neighbor, **d_neighbor;
  size_t neighborSize = NQ * sizeof(float *);
  
  CUDA_CALL(hipMalloc(&d_neighbor,neighborSize));
  neighbor = (float **)malloc(neighborSize);
  if(neighbor == NULL) {
    printf("Error allocating neighbor");
    exit(1);
  }

  cuNearestNeighbor<<<numberOfBlocks, threadsPerBlock>>>
    (d_S,d_SDim,d_Q,d_QBoxIdToCheck,d,d_neighbor);
  err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
             __FILE__,__LINE__);
      return EXIT_FAILURE;
  }


  CUDA_CALL(hipMemcpy(neighbor, d_neighbor, neighborSize, hipMemcpyDeviceToHost));

  /* Cleanup */
  CUDA_CALL(hipFree(d_Q));
  CUDA_CALL(hipFree(d_C));
  CUDA_CALL(hipFree(d_S));
  CUDA_CALL(hipFree(d_SDim));
  CUDA_CALL(hipFree(d_QBoxIdToCheck));
  free(Q);
  free(C);
  free(S);
  free(SDim);
  free(QBoxIdToCheck);
  
  return 0;
}